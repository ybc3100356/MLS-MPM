#include "hip/hip_runtime.h"
//
// Created by ybc on 2021/6/11.
//

#include "scene.h"
#include <cassert>
#include <hip/hip_cooperative_groups.h>

const int numBlock = 128;
const int numThread = 128;

__global__ void gpuCompute(Particle *particles, vec3 *grid_v, Real *grid_m) {
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    int threadId = int(grid.thread_rank());
    int totalThreadNum = int(grid.size());

    for (int step = 0; step < Scene::steps; step++) {
        // has to memset on each iter start
        if (threadId == 0) {
            memset(grid_v, 0, Scene::grid_v_size);
            memset(grid_m, 0, Scene::grid_m_size);
        }
        grid.sync();

        // p2g
//        assert(Scene::numParticles % totalThreadNum == 0);
        int pgRepeatTime = int(Scene::numParticles / totalThreadNum);
        for (int i = 0; i < pgRepeatTime; i++) {
            size_t idx = threadId * pgRepeatTime + i;
            if (idx > Scene::numParticles) continue;
            auto base = (ivec3) (particles[idx].position * Scene::inv_dx - 0.5f);
            auto fx = (particles[idx].position * Scene::inv_dx) - (vec3) base;
            // quadratic B-spline weights
            vec3 w[] = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                        0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                        0.5f * (fx - 0.5f) * (fx - 0.5f)};
            Real stress =
                    -Scene::dt * Scene::p_vol * (particles[idx].J - 1) * 4 * Scene::inv_dx * Scene::inv_dx * Scene::E;
            mat3 affine =
                    mat3(vec3(stress, 0, 0), vec3(0, stress, 0), vec3(0, 0, stress)) + Scene::p_mass * particles[idx].C;
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    for (int k = 0; k < 3; k++) {
                        auto offset = ivec3(i, j, k);
                        auto dpos = ((vec3) offset - fx) * Scene::dx;
                        auto weight = w[i][0] * w[j][1] * w[k][2];
                        auto index = base + offset;
                        if (!(index[0] < Scene::numGrid && index[1] < Scene::numGrid &&
                              index[2] < Scene::numGrid))
                            continue;
                        auto dv = weight * (Scene::p_mass * particles[idx].velocity + affine * dpos);
                        auto target_idx = index[0] * Scene::numGrid * Scene::numGrid + index[1] * Scene::numGrid + index[2];
                        atomicAdd(&(grid_v[target_idx][0]), dv[0]);
                        atomicAdd(&(grid_v[target_idx][1]), dv[1]);
                        atomicAdd(&(grid_v[target_idx][2]), dv[2]);
                        atomicAdd(&(grid_m[target_idx]), weight * Scene::p_mass);
                    }
                }
            }
        }
        grid.sync();

        // grid
        assert((Scene::numGrid * Scene::numGrid * Scene::numGrid) % totalThreadNum == 0);
        int gridRepeatTime = int((Scene::numGrid * Scene::numGrid * Scene::numGrid) / totalThreadNum);
        for (int i = 0; i < gridRepeatTime; i++) {
            size_t idx = threadId * gridRepeatTime + i;
            if (grid_m[idx] > 0) {
                auto inv_m = 1.0f / grid_m[idx];
                grid_v[idx] = inv_m * grid_v[idx];
                grid_v[idx][1] -= Scene::dt * 9.8;
                auto bound = 3;

                size_t i = idx / (Scene::numGrid * Scene::numGrid);
                size_t j = (idx / Scene::numGrid) % Scene::numGrid;
                size_t k = idx % Scene::numGrid;

                if (i < bound && grid_v[idx].x < 0)
                    grid_v[idx].x = 0;
                if (i > Scene::numGrid - bound && grid_v[idx].x > 0)
                    grid_v[idx].x = 0;

                if (j < bound && grid_v[idx].y < 0)
                    grid_v[idx].y = 0;
                if (j > Scene::numGrid - bound && grid_v[idx].y > 0)
                    grid_v[idx].y = 0;

                if (k < bound && grid_v[idx].z < 0)
                    grid_v[idx].z = 0;
                if (k > Scene::numGrid - bound && grid_v[idx].z > 0)
                    grid_v[idx].z = 0;
            }
        }
        grid.sync();

        // g2p
        for (int i = 0; i < pgRepeatTime; i++) {
            size_t idx = threadId * pgRepeatTime + i;
            if (idx > Scene::numParticles) continue;
            auto base = (ivec3) (particles[idx].position * Scene::inv_dx - 0.5f);
            auto fx = (particles[idx].position * Scene::inv_dx) - (vec3) base;
            // quadratic B-spline weights
            vec3 w[] = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                        0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                        0.5f * (fx - 0.5f) * (fx - 0.5f)};
            auto new_v = vec3(0);
            auto new_C = mat3(0);
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    for (int k = 0; k < 3; k++) {
                        auto offset = ivec3(i, j, k);
                        auto index = base + offset;
                        if (!(index[0] < Scene::numGrid && index[1] < Scene::numGrid &&
                              index[2] < Scene::numGrid))
                            continue;
                        auto weight = w[i][0] * w[j][1] * w[k][2];
                        auto dpos = ((vec3) offset - fx);
                        auto g_v = grid_v[index[0] * Scene::numGrid * Scene::numGrid + index[1] * Scene::numGrid + index[2]];
                        new_v += weight * g_v;
                        new_C += 4 * weight * outerProduct(g_v, dpos) * Scene::inv_dx;
                    }
                }
            }

            particles[idx].velocity = new_v;
            particles[idx].position += particles[idx].velocity * Scene::dt;
            particles[idx].J *= 1 + Scene::dt * (new_C[0][0] + new_C[1][1] + new_C[2][2]); // trace -- scale of volume
            particles[idx].C = new_C;
        }
        grid.sync();
    }
}

void Scene::update() {
    gpuUpdate();
}

void Scene::gpuUpdate() {
    hipMemcpy(particles_gpu, &particles[0], particles_size, hipMemcpyHostToDevice);
//    gpuCompute<<<numBlock, numThread>>>(particles_gpu, grid_v_gpu, grid_m_gpu);
    dim3 dimBlock(numThread, 1, 1);
    dim3 dimGrid(numBlock, 1, 1);
    void *kernelArgs[] = {
            (void *) &particles_gpu, (void *) &grid_v_gpu, (void *) &grid_m_gpu,
    };
    hipError_t code = hipLaunchCooperativeKernel((void *) gpuCompute, dimGrid, dimBlock, kernelArgs);
    if (code != hipSuccess) {
        fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
        exit(code);
    }
    hipDeviceSynchronize();
    hipMemcpy(&particles[0], particles_gpu, particles_size, hipMemcpyDeviceToHost);
}

void Scene::p2g() {
    // p2g
    for (auto &particle : particles) {
        auto base = (ivec3) (particle.position * inv_dx - 0.5f);
        auto fx = (particle.position * inv_dx) - (vec3) base;
        // quadratic B-spline weights
        vector<vec3> w = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                          0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                          0.5f * (fx - 0.5f) * (fx - 0.5f)};
        Real stress = -dt * p_vol * (particle.J - 1) * 4 * inv_dx * inv_dx * E;
        mat3 affine = mat3(vec3(stress, 0, 0), vec3(0, stress, 0), vec3(0, 0, stress)) + p_mass * particle.C;
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                for (int k = 0; k < 3; k++) {
                    auto offset = ivec3(i, j, k);
                    auto dpos = ((vec3) offset - fx) * dx;
                    auto weight = w[i][0] * w[j][1] * w[k][2];
                    auto index = base + offset;
                    assert (index[0] < numGrid && index[1] < numGrid && index[2] < numGrid);
                    grid_v[index[0]][index[1]][index[2]] += weight * (p_mass * particle.velocity + affine * dpos);
                    grid_m[index[0]][index[1]][index[2]] += weight * p_mass;
                }
            }
        }
    }
}

void Scene::g2p() {
    // g2p
    for (auto &particle : particles) {
        auto base = (ivec3) (particle.position * inv_dx - 0.5f);
        auto fx = (particle.position * inv_dx) - (vec3) base;
        // quadratic B-spline weights
        vector<vec3> w = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                          0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                          0.5f * (fx - 0.5f) * (fx - 0.5f)};

        auto new_v = vec3(0);
        auto new_C = mat3(0);
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                for (int k = 0; k < 3; k++) {
                    auto offset = ivec3(i, j, k);
                    auto index = base + offset;
                    assert (index[0] < numGrid && index[1] < numGrid && index[2] < numGrid);
                    auto weight = w[i][0] * w[j][1] * w[k][2];
                    auto dpos = ((vec3) offset - fx) * dx;
                    auto g_v = grid_v[index[0]][index[1]][index[2]];
                    new_v += weight * g_v;
                    new_C += 4 * weight * outerProduct(g_v, dpos) * inv_dx;
                }
            }
        }

        particle.velocity = new_v;
        particle.position += particle.velocity * dt;
        particle.J *= 1 + dt * (new_C[0][0] + new_C[1][1] + new_C[2][2]); // trace -- scale of volume
        particle.C = new_C;
    }
}

void Scene::gridCompute() {
    for (int i = 0; i < numGrid; i++) {
        for (int j = 0; j < numGrid; j++) {
            for (int k = 0; k < numGrid; k++) {
                if (grid_m[i][j][k] > 0) {
                    auto inv_m = 1.0f / grid_m[i][j][k];
                    grid_v[i][j][k] = inv_m * grid_v[i][j][k];
                    grid_v[i][j][k][1] -= dt * 9.8;
                    auto bound = 3;

                    if (i < bound && grid_v[i][j][k].x < 0)
                        grid_v[i][j][k].x = 0;
                    if (i > numGrid - bound && grid_v[i][j][k].x > 0)
                        grid_v[i][j][k].x = 0;

                    if (j < bound && grid_v[i][j][k].y < 0)
                        grid_v[i][j][k].y = 0;
                    if (j > numGrid - bound && grid_v[i][j][k].y > 0)
                        grid_v[i][j][k].y = 0;

                    if (k < bound && grid_v[i][j][k].z < 0)
                        grid_v[i][j][k].z = 0;
                    if (k > numGrid - bound && grid_v[i][j][k].z > 0)
                        grid_v[i][j][k].z = 0;
                }
            }
        }
    }
}

void Scene::subStep() {
    p2g();
    gridCompute();
    g2p();
}

void Scene::gpuInit() {
    hipMalloc((void **) &particles_gpu, particles_size);
    hipMalloc((void **) &grid_v_gpu, grid_v_size);
    hipMalloc((void **) &grid_m_gpu, grid_m_size);
}

void Scene::gpuFree() {
    hipFree(particles_gpu);
    hipFree(grid_v_gpu);
    hipFree(grid_m_gpu);
}

void Scene::render() {
    shader.use();
    shader.set("projection",
               glm::perspective(glm::radians(camera.Zoom), (float) SCR_WIDTH / (float) SCR_HEIGHT, 0.1f, 100.0f));
    shader.set("view", camera.getViewMatrix());
    glBindVertexArray(this->VAO);
    for (auto &particle : this->particles) {
        mat4 model = mat4(1.0f); // make sure to initialize matrix to identity matrix first
        model = glm::translate(model, particle.position);
        shader.set("model", model);
        shader.set("color", vec4(0, 0.5, 1, 1));
        glDrawArrays(GL_TRIANGLES, 0, 36);
    }
    glBindVertexArray(0);
}

Scene::Scene() : VAO(0),
                 grid_v(vector<vector<vector<vec3> > >
                                (numGrid, vector<vector<vec3>>(
                                        numGrid, vector<vec3>(
                                                numGrid, vec3())))),
                 grid_m(vector<vector<vector<Real> > >
                                (numGrid, vector<vector<Real>>(
                                        numGrid, vector<Real>(
                                                numGrid, 0)))),
                 particles(vector<Particle>(numParticles)) {
    gpuInit();

    GLuint VBO;
    GLfloat particleQuad[] = {
            -0.5f, -0.5f, -0.5f,
            0.5f, -0.5f, -0.5f,
            0.5f, 0.5f, -0.5f,
            0.5f, 0.5f, -0.5f,
            -0.5f, 0.5f, -0.5f,
            -0.5f, -0.5f, -0.5f,

            -0.5f, -0.5f, 0.5f,
            0.5f, -0.5f, 0.5f,
            0.5f, 0.5f, 0.5f,
            0.5f, 0.5f, 0.5f,
            -0.5f, 0.5f, 0.5f,
            -0.5f, -0.5f, 0.5f,

            -0.5f, 0.5f, 0.5f,
            -0.5f, 0.5f, -0.5f,
            -0.5f, -0.5f, -0.5f,
            -0.5f, -0.5f, -0.5f,
            -0.5f, -0.5f, 0.5f,
            -0.5f, 0.5f, 0.5f,

            0.5f, 0.5f, 0.5f,
            0.5f, 0.5f, -0.5f,
            0.5f, -0.5f, -0.5f,
            0.5f, -0.5f, -0.5f,
            0.5f, -0.5f, 0.5f,
            0.5f, 0.5f, 0.5f,

            -0.5f, -0.5f, -0.5f,
            0.5f, -0.5f, -0.5f,
            0.5f, -0.5f, 0.5f,
            0.5f, -0.5f, 0.5f,
            -0.5f, -0.5f, 0.5f,
            -0.5f, -0.5f, -0.5f,

            -0.5f, 0.5f, -0.5f,
            0.5f, 0.5f, -0.5f,
            0.5f, 0.5f, 0.5f,
            0.5f, 0.5f, 0.5f,
            -0.5f, 0.5f, 0.5f,
            -0.5f, 0.5f, -0.5f,
    };
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(particleQuad), particleQuad, GL_STATIC_DRAW);

    // Set mesh attributes
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(GLfloat), (GLvoid *) nullptr);
    glEnableVertexAttribArray(0);
    glBindVertexArray(0);

    camera = Camera(glm::vec3(0.25f, 0.9f, 1.7f), glm::vec3(0.0f, 1.0f, 0.0f), -75, -30);
}

Scene::~Scene() {
    glDeleteVertexArrays(1, &VAO);
    gpuFree();
}

void Scene::processInput(GLFWwindow *window, double deltaTime) {
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        camera.processKeyboard(FORWARD, deltaTime);
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        camera.processKeyboard(BACKWARD, deltaTime);
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        camera.processKeyboard(LEFT, deltaTime);
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        camera.processKeyboard(RIGHT, deltaTime);
    if (glfwGetKey(window, GLFW_KEY_U) == GLFW_PRESS)
        update();
}

void Scene::loadShader(const GLchar *vertexPath, const GLchar *fragmentPath) {
    shader.load(vertexPath, fragmentPath);
}

void Scene::processMouseMovement(float xOffset, float yOffset) {
    camera.processMouseMovement(xOffset, yOffset);
}

void Scene::processMouseScroll(float yOffset) {
    camera.processMouseScroll(yOffset);
}
