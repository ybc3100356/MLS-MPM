#include "hip/hip_runtime.h"
//
// Created by ybc on 2021/6/11.
//

#include "scene.h"
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

void Scene::update() {
    gpuUpdate();
}

const int blockNum = 32;
const int threadNum = 128;
constexpr Real e = 2.7182818284590452f;

__global__ void gpuCompute(Particle *particles, vec2 *grid_v, Real *grid_m) {
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    int threadId = int(grid.thread_rank());
    int totalThreadNum = int(grid.size());

    for (int step = 0; step < Scene::steps; step++) {
        // memset
        if (threadId == 0) {
            memset(grid_v, 0, Scene::grid_v_size);
            memset(grid_m, 0, Scene::grid_m_size);
        }
        grid.sync();

        // p2g
        assert(Scene::numParticles % totalThreadNum == 0);
        int pgRepeatTime = int(Scene::numParticles / totalThreadNum);
        for (int i = 0; i < pgRepeatTime; i++) {
            size_t idx = threadId * pgRepeatTime + i;
            auto &p = particles[idx];
            auto base = (ivec2) (p.position * Scene::inv_dx - 0.5f);
            auto fx = (p.position * Scene::inv_dx) - (vec2) base;
            // quadratic B-spline weights
            vec2 w[] = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                        0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                        0.5f * (fx - 0.5f) * (fx - 0.5f)};
            p.F = (mat2(1) + Scene::dt * p.C) * p.F;
            Real h = max(0.1f, min(5.0f, pow(e, 10 * (1.0f - p.Jp)))); // hardness
            if (p.material == Particle::Jelly) {
                h = 4;
            }
            Real mu = Scene::mu_0 * h;
            Real la = Scene::lambda_0 * h;
            if (p.material == Particle::Liquid) {
                mu = 0.0f;
            }

            //svd
            mat2 U, sig, V;
            mat2 R, S;
            Real x = p.F[0][0] + p.F[1][1], y = p.F[0][1] - p.F[1][0];
            Real scale = (1.0f / sqrt(x * x + y * y));
            Real cosx = x * scale;
            Real sinx = y * scale;
            R = mat2(cosx, sinx, -sinx, cosx);
            S = transpose(R) * p.F;
            Real c = 0, s = 0;
            Real s1 = 0, s2 = 0;
            if (fabs(S[1][0]) < 1e-5f) {
                c = 1;
                s = 0;
                s1 = S[0][0];
                s2 = S[1][1];
            } else {
                Real tao = 0.5f * (S[0][0] - S[1][1]);
                Real w = sqrt(tao * tao + S[1][0] * S[1][0]);
                Real t = 0;
                if (tao > 0) {
                    t = S[1][0] / (tao + w);
                } else {
                    t = S[1][0] / (tao - w);
                }
                c = 1.0f / sqrt(t * t + 1);
                s = -t * c;
                s1 = c * c * S[0][0] - 2 * c * s * S[1][0] + s * s * S[1][1];
                s2 = s * s * S[0][0] + 2 * c * s * S[1][0] + c * c * S[1][1];
            }
            if (s1 < s2) {
                Real tmp = s1;
                s1 = s2;
                s2 = tmp;
                V = mat2(-s, -c, c, -s);
            } else {
                V = mat2(c, -s, s, c);
            }
            U = R * V;
            sig = mat2(s1, 0, 0, s2);

            Real J = 1.0f;
            for (int d = 0; d < 2; d++) {
                auto newSig = sig[d][d];
                if (p.material == Particle::Snow) {
                    newSig = min(max(sig[d][d], 1 - 2.5e-2f), 1 + 1.5e-3f);
                }
                p.Jp *= sig[d][d] / newSig;
                sig[d][d] = newSig;
                J *= newSig;
            }
            if (p.material == Particle::Liquid) {
                p.F = mat2(1) * sqrt(J);
            } else if (p.material == Particle::Snow) {
                p.F = U * sig * transpose(V);
            }
            auto stress = 2 * mu * (p.F - U * transpose(V)) * transpose(p.F) + mat2(1) * la * J * (J - 1);
            stress = stress * (-Scene::dt * Scene::p_vol * 4 * Scene::inv_dx * Scene::inv_dx);
            auto affine = stress + Scene::p_mass * p.C;
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    auto offset = ivec2(i, j);
                    auto dpos = ((vec2) offset - fx) * Scene::dx;
                    auto weight = w[i][0] * w[j][1];
                    auto index = base + offset;
                    if (!(index[0] < Scene::numGrid && index[1] < Scene::numGrid)) continue;
                    auto dv = weight * (Scene::p_mass * p.velocity + affine * dpos);
                    auto target_idx = index[0] * Scene::numGrid + index[1];
                    atomicAdd(&(grid_v[target_idx][0]), dv[0]);
                    atomicAdd(&(grid_v[target_idx][1]), dv[1]);
                    atomicAdd(&(grid_m[target_idx]), weight * Scene::p_mass);
                }
            }
        }
        grid.sync();

        // grid
        assert((Scene::numGrid * Scene::numGrid) % totalThreadNum == 0);
        int gridRepeatTime = int((Scene::numGrid * Scene::numGrid) / totalThreadNum);
        for (int i = 0; i < gridRepeatTime; i++) {
            size_t idx = threadId * gridRepeatTime + i;
            if (grid_m[idx] > 0) {
                auto inv_m = 1.0f / grid_m[idx];
                grid_v[idx] = inv_m * grid_v[idx];
                grid_v[idx][1] -= Scene::dt * 30;
                auto bound = 3;
                size_t i = idx / Scene::numGrid;
                size_t j = idx % Scene::numGrid;
                if (i < bound && grid_v[idx][0] < 0)
                    grid_v[idx][0] = 0;
                if (i > Scene::numGrid - bound && grid_v[idx][0] > 0)
                    grid_v[idx][0] = 0;
                if (j < bound && grid_v[idx][1] < 0)
                    grid_v[idx][1] = 0;
                if (j > Scene::numGrid - bound && grid_v[idx][1] > 0)
                    grid_v[idx][1] = 0;
            }
        }
        grid.sync();

        // g2p
        for (int i = 0; i < pgRepeatTime; i++) {
            size_t idx = threadId * pgRepeatTime + i;
            auto base = (ivec2) (particles[idx].position * Scene::inv_dx - 0.5f);
            auto fx = (particles[idx].position * Scene::inv_dx) - (vec2) base;
            // quadratic B-spline weights
            vec2 w[] = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                        0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                        0.5f * (fx - 0.5f) * (fx - 0.5f)};
            auto new_v = vec2(0, 0);
            auto new_C = mat2(0);
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    auto offset = ivec2(i, j);
                    auto index = base + offset;
                    if (!(index[0] < Scene::numGrid && index[1] < Scene::numGrid)) continue;
                    auto weight = w[i][0] * w[j][1];
                    auto dpos = ((vec2) offset - fx) * Scene::dx;
                    auto g_v = grid_v[index[0] * Scene::numGrid + index[1]];
                    new_v += weight * g_v;
                    new_C += 4 * weight * outerProduct(g_v, dpos) * Scene::inv_dx;
                }
            }

            particles[idx].velocity = new_v;
            particles[idx].C = new_C;
            particles[idx].position += particles[idx].velocity * Scene::dt; // boundary
        }
        grid.sync();
    }
}

void Scene::gpuInit() {
    hipMalloc((void **) &particles_gpu, particles_size);
    hipMalloc((void **) &grid_v_gpu, grid_v_size);
    hipMalloc((void **) &grid_m_gpu, grid_m_size);
}

void Scene::gpuFree() {
    hipFree(particles_gpu);
    hipFree(grid_v_gpu);
    hipFree(grid_m_gpu);
}

void Scene::gpuUpdate() {
    hipMemcpy(particles_gpu, &particles[0], particles_size, hipMemcpyHostToDevice);
//    gpuCompute<<<blockNum, threadNum>>>(particles_gpu, grid_v_gpu, grid_m_gpu);
//    hipError_t code = hipPeekAtLastError();
    dim3 dimBlock(threadNum, 1, 1);
    dim3 dimGrid(blockNum, 1, 1);
    void *kernelArgs[] = {
            (void *) &particles_gpu, (void *) &grid_v_gpu, (void *) &grid_m_gpu,
    };
    hipError_t code = hipLaunchCooperativeKernel((void *) gpuCompute, dimGrid, dimBlock, kernelArgs);
    if (code != hipSuccess) {
        fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
        exit(code);
    }
    hipDeviceSynchronize();
    hipMemcpy(&particles[0], particles_gpu, particles_size, hipMemcpyDeviceToHost);
}

void Scene::subStep() {
    p2g();
    gridCompute();
    g2p();
}

void Scene::render() {
    shader.use();
    for (auto &particle : this->particles) {
        shader.set("offset", particle.position);
        shader.set("color", particle.color);
        glBindVertexArray(this->VAO);
        glDrawArrays(GL_TRIANGLES, 0, 6);
        glBindVertexArray(0);
    }
}

Scene::Scene(const Shader &shader) : shader(const_cast<Shader &>(shader)), VAO(0),
                                     grid_v(vector<vector<vec2>>(numGrid, vector<vec2>(numGrid, vec2(0, 0)))),
                                     grid_m(vector<vector<Real>>(numGrid, vector<Real>(numGrid, 0))) {
    for (int i = 0; i < numParticlesPerObject; i++)
        particles.emplace_back(vec2(0.35, 0.45), vec4(237 / 255., 85 / 255., 59 / 255., 1),
                               Particle::Jelly);
    for (int i = 0; i < numParticlesPerObject; i++)
        particles.emplace_back(vec2(0.45, 0.65), vec4(242 / 255., 177 / 255., 52 / 255., 1),
                               Particle::Liquid);
    for (int i = 0; i < numParticlesPerObject; i++)
        particles.emplace_back(vec2(0.55, 0.85), vec4(6 / 255., 133 / 255., 135 / 255., 1),
                               Particle::Snow);

    gpuInit();

    GLuint VBO;
    GLfloat particleQuad[] = {
            0.0f, 1.0f,
            1.0f, 0.0f,
            0.0f, 0.0f,

            0.0f, 1.0f,
            1.0f, 1.0f,
            1.0f, 0.0f,
    };
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(particleQuad), particleQuad, GL_STATIC_DRAW);

    // Set mesh attributes
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(GLfloat), (GLvoid *) nullptr);
    glEnableVertexAttribArray(0);
    glBindVertexArray(0);
}

Scene::~Scene() {
//    glDeleteVertexArrays(1, &VAO);
    gpuFree();
}

void Scene::p2g() {
    // p2g
    for (auto &particle : particles) {
        auto base = (ivec2) (particle.position * inv_dx - 0.5f);
        auto fx = (particle.position * inv_dx) - (vec2) base;
        // quadratic B-spline weights
        vector<vec2> w = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                          0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                          0.5f * (fx - 0.5f) * (fx - 0.5f)};
        Real stress = -dt * p_vol * (particle.Jp - 1) * 4 * inv_dx * inv_dx * E;
        auto affine = mat2(vec2(stress, 0), vec2(0, stress)) + p_mass * particle.C;
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                auto offset = ivec2(i, j);
                auto dpos = ((vec2) offset - fx) * dx;
                auto weight = w[i][0] * w[j][1];
                auto index = base + offset;
                assert (index[0] < numGrid && index[1] < numGrid);
                grid_v[index[0]][index[1]] += weight * (p_mass * particle.velocity + affine * dpos);
                grid_m[index[0]][index[1]] += weight * p_mass;
            }
        }
    }
}

void Scene::g2p() {
    // g2p
    for (auto &particle : particles) {
        auto base = (ivec2) (particle.position * inv_dx - 0.5f);
        auto fx = (particle.position * inv_dx) - (vec2) base;
        // quadratic B-spline weights
        vector<vec2> w = {0.5f * ((1.5f - fx) * (1.5f - fx)),
                          0.75f - ((fx - 1.0f) * (fx - 1.0f)),
                          0.5f * (fx - 0.5f) * (fx - 0.5f)};

        auto new_v = vec2(0, 0);
        auto new_C = mat2();
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                auto offset = ivec2(i, j);
                auto index = base + offset;
                assert (index[0] < numGrid && index[1] < numGrid);
                auto weight = w[i][0] * w[j][1];
                auto dpos = ((vec2) offset - fx) * dx;
                auto g_v = grid_v[index[0]][index[1]];
                new_v += weight * g_v;
                new_C += 4 * weight * outerProduct(g_v, dpos) * inv_dx;
            }
        }

        particle.velocity = new_v;
        particle.position += particle.velocity * dt; // boundary
        particle.Jp *= 1 + dt * (new_C[0][0] + new_C[1][1]); // trace -- scale of volume
        particle.C = new_C;
    }
}

void Scene::gridCompute() {
    for (int i = 0; i < grid_m.size(); i++) {
        for (int j = 0; j < grid_m[0].size(); j++) {
            if (grid_m[i][j] > 0) {
                auto inv_m = 1.0f / grid_m[i][j];
                grid_v[i][j] = inv_m * grid_v[i][j];
                grid_v[i][j][1] -= dt * 9.8;
                auto bound = 3;
                if (i < bound && grid_v[i][j][0] < 0)
                    grid_v[i][j][0] = 0;
                if (i > numGrid - bound && grid_v[i][j][0] > 0)
                    grid_v[i][j][0] = 0;
                if (j < bound && grid_v[i][j][1] < 0)
                    grid_v[i][j][1] = 0;
                if (j > numGrid - bound && grid_v[i][j][1] > 0)
                    grid_v[i][j][1] = 0;
            }
        }
    }
}
